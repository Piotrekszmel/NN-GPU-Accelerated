#include "hip/hip_runtime.h"
#include "utils.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

float randFloat(float min, float max)
{
   return min + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (min - max)));
}

int randInt(int min, int max)
{
   return min + (rand() % static_cast<int>(max - min + 1));
}
