#include "hip/hip_runtime.h"
#include "tensor.cuh"

Tensor::Tensor(int sizeX, int sizeY)
{
    m_sizeX = sizeX;
    m_sizeY = sizeY;

    if (m_sizeX && m_sizeY)
    {
        hipMalloc((void**)&m_devData, m_sizeX * m_sizeY * sizeof(float));
    }
    else
    {
        m_devData = NULL;
    }
}

Tensor::Tensor(float* data, int sizeX, int sizeY, DataType dataType)
{
    m_sizeX = sizeX;
    m_sizeY = sizeY;
    if (dataType == HOST)
    {
        if (m_sizeX && m_sizeY)
        {
            gpuErrCheck(hipMalloc((void**)&m_devData, m_sizeX * m_sizeY * sizeof(float)));
            gpuErrCheck(hipMemcpy(m_devData, data, m_sizeX * m_sizeY * sizeof(float), hipMemcpyHostToDevice));
        }
        else
        {
            m_devData = NULL;
        }
    }
    else if (dataType == DEVICE)
    {
        m_devData = data;
        m_sizeX = sizeX;
        m_sizeY = sizeY;
    }
    else
    {
        printf("Wrong DataType\n");
    }
}

Tensor::~Tensor()
{
    hipFree(m_devData);
}

int Tensor::getSize(Axis ax)
{
    if (ax == X)
        return m_sizeX;
    else if (ax == Y)
        return m_sizeY;
    return -1;
}

float* Tensor::getDeviceData()
{
    return m_devData;
}

void Tensor::fetchDeviceData(float** hostData)
{
    *hostData = (float*)malloc(m_sizeX * m_sizeY * sizeof(float));
    gpuErrCheck(hipMemcpy(hostData,
                           m_devData,
                           m_sizeX * m_sizeX * sizeof(float),
                           hipMemcpyDeviceToHost));
}

__global__ void add_kernel(float* a, float* b, int sizeX, int sizeY)
{
    int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int y_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_idx < sizeX && y_idx < sizeY)
    {
        a[y_idx * sizeX + x_idx] += b[y_idx * sizeX + x_idx];
    }
}

void Tensor::add(Tensor& tensor)
{
    if (m_sizeX != tensor.getSize(X) || m_sizeY != tensor.getSize(Y))
    {
        printf("Tensors have to have the same shapes.\nTensor1: [%d, %d]\nTensor2: [%d, %d]\n",
               m_sizeX, m_sizeY, tensor.getSize(X), tensor.getSize(Y));
        exit(1);
    }

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(1,1,1);
    add_kernel<<<numBlocks, threadsPerBlock>>>(getDeviceData(), tensor.getDeviceData(), m_sizeX, m_sizeY);
}
