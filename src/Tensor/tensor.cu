#include "tensor.cuh"

Tensor::Tensor(int sizeX, int sizeY)
{
    m_sizeX = sizeX;
    m_sizeY = sizeY;

    if (m_sizeX && m_sizeY)
    {
        hipMalloc((void**)&m_devData, m_sizeX * m_sizeY * sizeof(float));
    }
    else
    {
        m_devData = NULL;
    }
}

Tensor::Tensor(float* hostData, int sizeX, int sizeY)

    m_sizeX = sizeX;
    m_sizeY = sizeY;

    if (m_sizeX && m_sizeY)
    {
        hipMalloc((void**)&m_devData, m_sizeX * m_sizeY * sizeof(float));
        hipMemcpy(m_devData, hostData, m_sizeX * m_sizeY * sizeof(float), cudeMemcpyHostToDevice);
        //add error check
    }
    else
    {
        m_devData = NULL;
    }
}